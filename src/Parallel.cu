#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <chrono>
#include <omp.h>
#include <fstream>
#include <string>
using namespace std;
using namespace chrono;

int numberOfEntries(string filename)
{
	ifstream file(filename);
	if (file.is_open())
	{
		string line;
		int i = 0;
		while (getline(file, line))
			i++;
		file.close();
		return i;
	}
	else
	{
		cout << "Couldn't find file " << filename << " !" << endl;
		return -1;
	}
}
void readFileToArray(string filename, double* arr, int N)
{
	ifstream file(filename);
	if (file.is_open())
	{
		string line;
		int i = 0;
		while (getline(file, line) && i<N)
		{
			arr[i] = atof(line.c_str());
			i++;
		}
		file.close();
	}
	else
	{
		cout << "Couldn't find file " << filename << " !" << endl;
	}
}

__global__ void calculateRanks(double* r_next, double* P, double* r, double* c, int N)
{
	int	i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < N)
	{
		r_next[i] = c[i];
		for (int j = 0; j < N; j++)
			r_next[i] += P[i*N + j] * r[j];
	}
}
__global__ void updateValues(double* r, double* r_next, int N)
{
	int	i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<N)
		r[i] = r_next[i];
}



int main(int argc, char *argv[])
{
	//Default settings
	int N = -1;
	int N2 = N*N;
	double a = 0.2;
	int maxIter = 50;
	double epsilon = pow(10, -5);
	int OpenMPThreads = 1;
	string filename = "";

	if (argc < 3)
	{
		cout << "Need at least -file argument !" << endl;
		return -1;
	}

	//Override with command line arguments
	for (int i = 1; i < argc; i++)
	{
		if (strcmp(argv[i], "-file") == 0)
		{
			filename = argv[i + 1];
			i++;
		}
		else if (strcmp(argv[i], "-iter") == 0)
		{
			maxIter = atoi(argv[i + 1]);
			i++;
		}
		else if (strcmp(argv[i], "-epsilon") == 0)
		{
			epsilon = pow(10, atoi(argv[i + 1]));
			i++;
		}
		else if (strcmp(argv[i], "-threads") == 0)
		{
			OpenMPThreads = atoi(argv[i + 1]);
			i++;
		}
	}

	omp_set_num_threads(OpenMPThreads);

	if (filename == "")
	{
		cout << "Need a link matrix file !" << endl;
		return -1;
	}

	N2 = numberOfEntries(filename);

	if (N2 == -1)
	{
		cout << "Error reading a file !" << endl;
		return -1;
	}

	N = sqrt(N2);

	//Show arguments
	cout << N << "x" << N << " matrix " << endl;
	cout << "Maximum iteration count = " << maxIter << endl;
	cout << "Epsilon = " << epsilon << endl;
	cout << "OpenMP threads = " << OpenMPThreads << endl;

	//Memory allocations
	double* c = new double[N];
	double* r = new double[N];
	double* r_next = new double[N];
	double* P = new double[N2];

	//Init arrays
	#pragma omp parallel for
	for (int i = 0; i < N; i++)
	{
		c[i] = 1.0 - a;
		r[i] = 1.0 / N;
		r_next[i] = 0.0;
	}
	//Override link matrix with file
	readFileToArray(filename, P, N2);

	//Damping factor
	#pragma omp parallel for
	for (int i = 0; i < N2; i++)
		P[i] *= a;

	//GPU memory allocations
	double* c_GPU, *r_GPU, *r_next_GPU, *P_GPU;
	hipMalloc((void**)&c_GPU, sizeof(double)*N);
	hipMalloc((void**)&r_GPU, sizeof(double)*N);
	hipMalloc((void**)&r_next_GPU, sizeof(double)*N);
	hipMalloc((void**)&P_GPU, sizeof(double)*N2);
	//Copy to GPU
	hipMemcpy(c_GPU, c, sizeof(double)*N, hipMemcpyHostToDevice);
	hipMemcpy(r_GPU, r, sizeof(double)*N, hipMemcpyHostToDevice);
	hipMemcpy(r_next_GPU, r_next, sizeof(double)*N, hipMemcpyHostToDevice);
	hipMemcpy(P_GPU, P, sizeof(double)*N2, hipMemcpyHostToDevice);


	//Start calculating
	int n = 0;
	auto startTime = high_resolution_clock::now();
	while (true)
	{
		//Calculate r_next
		int threads = 1024;
		int blocks = N / 1024;
		if (blocks == 0)
			blocks = 1;
		calculateRanks << <blocks, threads >> > (r_next_GPU, P_GPU, r_GPU, c_GPU, N);
		hipDeviceSynchronize();
		n++;

		//Calculate the stopping condition
		hipMemcpy(r, r_GPU, sizeof(double)*N, hipMemcpyDeviceToHost);
		hipMemcpy(r_next, r_next_GPU, sizeof(double)*N, hipMemcpyDeviceToHost);
		double z = 0.0;
		#pragma omp parallel for reduction(+:z)
		for (int i = 0; i < N; i++)
			z += abs(r_next[i] - r[i]);

		//Switch pointers
		updateValues << <N, 1 >> > (r_GPU, r_next_GPU, N);
		hipDeviceSynchronize();

		//Check if should stop
		if (z <= epsilon || maxIter == n)
			break;
	}

	//Print stats
	auto endTime = high_resolution_clock::now();
	auto duration = endTime - startTime;
	hipMemcpy(r, r_GPU, sizeof(double)*N, hipMemcpyDeviceToHost);
	cout << "Calculation done in " << (duration_cast<milliseconds>(duration).count()) << " ms!" << endl;
	cout << "Number of iterations calculated: " << n << endl;

	ofstream out(filename + "_Parallel_" + to_string(OpenMPThreads) + " _PageRanks.txt");
	if (out.is_open())
	{
		for (int i = 0; i < N; i++)
			out << r[i] << endl;
		out.close();
	}
	else
		cout << "Couldn't write result to file !" << endl;


	//Free host memory
	delete c;
	delete r;
	delete r_next;
	delete P;
	//Free device memory
	hipFree(c_GPU);
	hipFree(r_GPU);
	hipFree(r_next_GPU);
	hipFree(P_GPU);

	return 0;
}